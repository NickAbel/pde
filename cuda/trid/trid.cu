#include "hip/hip_runtime.h"
//
// Program to perform Backward Euler time-marching on a 1D grid
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>

////////////////////////////////////////////////////////////////////////
// include kernel function
////////////////////////////////////////////////////////////////////////

#include <trid_kernel.h>

////////////////////////////////////////////////////////////////////////
// declare Gold routine
////////////////////////////////////////////////////////////////////////

void gold_trid(int, int, float*, float*);

////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////

int main(int argc, const char **argv){

  int    NX = 16, niter = 5;

  float *h_u, *h_v, *h_c, *d_u;

  // initialise card

  findCudaDevice(argc, argv);

  // allocate memory on host and device

  h_u = (float *)malloc(sizeof(float)*NX);
  h_v = (float *)malloc(sizeof(float)*NX);
  h_c = (float *)malloc(sizeof(float)*NX);

  checkCudaErrors( hipMalloc((void **)&d_u, sizeof(float)*NX) );

  // GPU execution

  for (int i=0; i<NX; i++) h_u[i] = 0.0f;
  h_u[3]=h_u[6]=0.1;
  h_u[4]=h_u[5]=0.2;
  checkCudaErrors( hipMemcpy(d_u, h_u, sizeof(float)*NX,
                              hipMemcpyHostToDevice) );

  GPU_trid<<<1, NX>>>(NX, niter, d_u);

  checkCudaErrors( hipMemcpy(h_u, d_u, sizeof(float)*NX,
                              hipMemcpyDeviceToHost) );


  // CPU execution

  for (int i=0; i<NX; i++) h_v[i] = 1.0f;

  gold_trid(NX, niter, h_v, h_c);


  // print out array

  for (int i=0; i<NX; i++) {
    printf(" %d  %f  %f  %f \n",i,h_u[i],h_v[i], abs(h_u[i]-h_v[i]));
  }

 // Release GPU and CPU memory

  checkCudaErrors( hipFree(d_u) );

  free(h_u);
  free(h_v);
  free(h_c);

}
