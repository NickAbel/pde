#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>


__global__ void WaveEq(float *d_mm1, float *d_m, float *d_mp1, float s,
		       float T, float dt, float cfl)
{
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  d_m[i]=d_mp1[i];
  float t=0.0;
  while (t < T) {
    t=dt+t;
    d_mm1[i]=d_m[i];
    d_m[i]=d_mp1[i];
    __syncthreads();
    if (i>0 && i<(blockDim.x-1)) {
      d_mp1[i]=2*d_m[i]-d_mm1[i]+s*(d_m[i-1]-2*d_m[i]+d_m[i+1]);
    }
  }
}

int main(int argc, char** argv)
{
  const int n=100;
  const int BYTES=n*sizeof(float);
  float h_mm1[n];
  float h_m[n];
  float h_mp1[n];
  float c=1.0;
  float T=1.0;
  float dx=0.1;
  float dt=dx/c;
  float cfl=c*dt/dx;
  float s=cfl*cfl;
  //initialize arrays
  for (int i=0;i<n;i++)
    {
      h_mm1[i]=0.0;
      h_m[i]=0.0;
      h_mp1[i]=0.0;
    }
  h_mp1[48]=0.1f;
  h_mp1[50]=0.1f;
  h_mp1[49]=0.2f;

  //declare GPU memory pointers
  float* d_mm1;
  float* d_m;
  float* d_mp1;

  //allocate memory on the device
  hipMalloc((void**)&d_mm1,BYTES);
  hipMalloc((void**)&d_m,BYTES);
  hipMalloc((void**)&d_mp1,BYTES);

  //transfer the array to the GPU
  //destination, source, size, method
  hipMemcpy(d_mm1,h_mm1,BYTES,hipMemcpyHostToDevice);
  hipMemcpy(d_m,h_m,BYTES,hipMemcpyHostToDevice);
  hipMemcpy(d_mp1,h_mp1,BYTES,hipMemcpyHostToDevice);

  //launch the kernel
  WaveEq<<<1,n>>>(d_mm1,d_m,d_mp1,s,T,dt,cfl);
  hipDeviceSynchronize();

  //copy the results back onto the device
  //destination, source, size, method
  hipMemcpy(h_mm1,d_mm1,BYTES,hipMemcpyDeviceToHost);
  hipMemcpy(h_m,d_m,BYTES,hipMemcpyDeviceToHost);
  hipMemcpy(h_mp1,d_mp1,BYTES,hipMemcpyDeviceToHost);

  for (int i=0;i<n;i++) 
    {
      printf("%d \t %f",i,h_mp1[i]);
      printf("\n");
    }

  printf("\n");

  //free memory previously allocated on the device
  hipFree(d_mm1);
  hipFree(d_m);
  hipFree(d_mp1);
}